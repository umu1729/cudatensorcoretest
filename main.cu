
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <mma.h>
using namespace nvcuda;


#define WARP_SIZE 32



__global__ void cuda_tensor(float *a, float *b, float *c){

    __shared__ __align__(4) hip_bfloat16 s_a[16*16];
    __shared__ __align__(4) hip_bfloat16 s_b[16*16];
    __shared__ __align__(4) float s_c[16*16];
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    #pragma unroll
    for (int i=0; i<16*16/WARP_SIZE; i++){
        int idx = i*WARP_SIZE + tid;
        s_a[idx] = __float2bfloat16(a[idx]);
        s_b[idx] = __float2bfloat16(b[idx]);
    }
    __syncthreads();
    
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> fA;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, __hip_bfloat16, wmma::row_major> fB;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> fC;
    wmma::fill_fragment(fC, 0.0f);
    
    wmma::load_matrix_sync(fA, s_a, 16);
    wmma::load_matrix_sync(fB, s_b, 16);
    wmma::mma_sync(fC, fA, fB, fC);
    wmma::store_matrix_sync(s_c, fC, 16, wmma::mem_row_major);

    #pragma unroll
    for (int i=0; i<16*16/WARP_SIZE; i++){
        int idx = i*WARP_SIZE + tid;
        c[idx] = s_c[idx];
    }
    __syncthreads();
}

int main() {

    float c_a[16*16];
    float c_b[16*16];
    float c_c[16*16];
    
    for (int i=0; i<16*16; i++) c_a[i] = i;
    for (int i=0; i<16*16; i++) c_b[i] = i*(i%3);
    for (int i=0; i<16*16; i++) c_c[i] = 0;

    float *d_a, *d_b, *d_c;
    hipMalloc( (void**)&d_a, 16*16*sizeof(float) );
    hipMalloc( (void**)&d_b, 16*16*sizeof(float) );
    hipMalloc( (void**)&d_c, 16*16*sizeof(float) );
    hipMemcpy(d_a, c_a, 16*16*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, c_b, 16*16*sizeof(float), hipMemcpyHostToDevice);
    cuda_tensor<<<1, WARP_SIZE>>>(d_a, d_b, d_c); 
    hipDeviceSynchronize();
    hipMemcpy(c_c, d_c, 16*16*sizeof(float), hipMemcpyDeviceToHost);
    
    
    float chk[16*16];
    for (int i=0; i<16*16; i++) chk[i]=0;
    for (int i=0; i<16; i++) for (int j=0; j<16; j++) for (int k=0; k<16; k++)
        chk[i*16+j] += c_a[i*16+k] * c_b[k*16+j];
    
    float eps = 1e-5;
    bool flag = true;
    printf("host tensor-core diff\n");
    for (int i=0; i<16*16; i++) {
        printf("%f %f %f \n", chk[i], c_c[i], chk[i]- c_c[i]);
        flag &= (chk[i]- c_c[i]) < eps;
        flag &= -eps < (chk[i]- c_c[i]);
    }
    printf("\x1b[31m");
    if (flag) printf("******** OK! ********\n"); else printf ("******** WRONG... ********\n");
    printf("\x1b[0m");
    
    return 0;
}